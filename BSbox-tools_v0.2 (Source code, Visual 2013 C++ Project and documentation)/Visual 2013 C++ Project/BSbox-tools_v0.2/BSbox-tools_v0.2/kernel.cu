////////////////////////////////////////////////////////////////////////////
//
// Copyright @2018 Dusan and Iliya.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////
//Example for using GPU BoolSPLG library procedures for Boolean function  
////////////////////////////////////////////////////////////////////////////

//System includes
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <string>
#include <sstream>
#include <fstream>
#include <time.h>

// CUDA runtime
#include "hip/hip_runtime.h"
#include ""

//Main Library header file
#include <BoolSPL02\BoolSPLG_v02.cuh> //#include "BoolSPLG_v0.cuh" 

//Output file 
std::ofstream myfile;

//Declaration strings for input and output files names
std::string infile_exampl = "infile_exampl";
std::string outfile_exampl = "outfile_exampl";


//@Global Time computation variables
double elapsedTime_computation_part=0, elapsedTime_computation=0;

//@@Global Size variable
int size = 0;

//@@Global Bool variables
int sizeBool;

//host vectors for Boolean function
int  *host_Bool_TT, *host_Bool_PTT, *host_GPUbool_rez, *walshvec_cpu, *rf_cpu, *anf_cpu;

//host vectors for Boolean Bitwise function
unsigned long long int *host_NumIntVecTT, *host_NumIntVecANF;
int *host_max_values_AD;

int *walshvec_gpu;

//device bool vectors
int *device_Bool, *device_Bool_TT, *device_Bool_PTT, *device_Bool_rez;

//Bitwise device vectors
unsigned long long int *device_NumIntVecTT, *device_NumIntVecANF;
int *device_max_values_AD;

//host vectors for S-box
int *LAT, *ACT, *ANF, *CF, *DDT, *PTT, *TT, *t, *WHT, *AC_all, *ANF_S, *DEG_ALL, *delta_ALL, *binary_num;

//device S-box vectors
int *device_Sbox, *device_CF, *device_LAT, *device_ACT, *device_ANF, *device_DDT;

//Bitwise ANF computation
unsigned long long int *NumIntVecTT, *NumIntVecANF; // , *mack_vec_Int;

//@Declaration of host - device vector for Bitwise ANF computation (S-box)
unsigned long long int *device_NumIntVecCF, *host_NumIntVecCF;
int *host_CF, *host_max_values, *device_Vec_max_values;


//@@S-box variables
int sizeSbox, binary = 0;

int *SboxElemet, *BoolElemet;


//@@Variable borders
int nl_gpuBorder = 0, AC_gpuBorder = 0, deg_gpuBorder = 0, delta_gpuBorder = 0;
int nl_cpuBorder = 0, AC_cpuBorder = 0, deg_cpuBorder = 0, delta_cpuBorder = 0;

//global variable
int choice1, choice2;

//function menu
void menu();
void mainMenu();

void menuCPU();
void mainMenuCPU();

void CUDA_Properties();
void CUDA_PropertiesMenu();

void CPU_boolMenu();
void CPU_bool();

void GPU_boolMenu();
void GPU_bool();

void CPU_SboxMenu();
void CPU_Sbox();

void GPU_SboxMenu();
void GPU_Sbox();

void findCPU_boolMenu();
void findCPU_bool();

void findGPU_boolMenu();
void findGPU_bool();

void findCPU_sboxMenu();
void findCPU_sbox();

void findGPU_sboxMenu();
void findGPU_sbox();

//Help Heder file - Input Output file
#include "InputOutputFiles.h"

//Help Header file - Time functions
#include "TimeHeader.h"

//Help Heder file - CPU computing boolean functions properties
#include "func_Boolean_CPU.h"
#include "funct_Sbox_CPU.h"

//Heder file 2D DynamicArray
#include "2D_DynamicArray.h"

//Help Heder file boolean computation functions
#include "HelpBoolFunctionGPU.h"
#include "HelpBoolFunctionCPU.h"

#include "HelpFindBoolFunctionCPU.h"
#include "HelpFindBoolFunctionGPU.h"


//Help Heder file S-box computation functions
#include "HelpSboxFunctionCPU.h"
#include "HelpSboxFunctionGPU.h"

#include "HelpFindSboxFunctionCPU.h"
#include "HelpFindSboxFunctionGPU.h"

//Header file that contain S-box generation
#include "GenBoolean.h"
#include "GenSboxes.h"

// === Menu Heders files ===
#include "MainMenuCPU.h"
#include "MainMenu.h"

//Menu Heders CUDA Properties
#include "CUDA_Properties_menu.h"

//Menu Heder CPU - GPU Bool Properties
#include "ComputePropMenuBoolGPU.h"
#include "ComputePropMenuBoolCPU.h"

//Menu Heder CPU - GPU S-box Properties
#include "ComputePropMenuSboxCPU.h"
#include "ComputePropMenuSboxGPU.h"

#include "FindMenuBoolCPU.h"
#include "FindMenuBoolGPU.h"
#include "FindMenuSboxCPU.h"
#include "FindMenuSboxGPU.h"

//Help Heder file - Check for hardware Requires
#include "CheckRequires.h"

using namespace std;

int main()
{
 
	printf("\n   Current release: v0.2 \n");

	cout << "\n==========================================================\n";
	//Function Check Hardware Requires
	BoolSPLGMinimalRequires1();
	cout << "\n==========================================================\n";

	//Function create Output files
	CreateOutputFile();

	//Function Open Main Menu
	menu();

    return 0;
}

